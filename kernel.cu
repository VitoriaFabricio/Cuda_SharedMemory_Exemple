
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#define filterWidth 3
#define filterHeight 3
#define TILE_SIZE 32

void checkCUDAError(hipError_t cudaStatus, const char* errorMessage) {
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "%s: %s\n", errorMessage, hipGetErrorString(cudaStatus));
        exit(1);
    }
}

__global__ void filter(const unsigned int* input_image, unsigned int* output_image, unsigned int height, unsigned int width) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Indices shared_block
    int shared_x = threadIdx.x + 1;
    int shared_y = threadIdx.y + 1;

    // ThreadIdx.x and threadIdx.y  (0-31)
    // If threadIdx.x is '0' shared_x is '1'

    __shared__ unsigned int shared_block[(32+2) * (32+2)];

    // Center 
    for (int i = 0; i < 32; i++) {
        if (y < height) {
            shared_block[(shared_y) * (34) + shared_x] = input_image[y * 1920 + x];
        }
    }

    // Corners

    // Top-left corner
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        if(x > 0 && y > 0){
              shared_block[0] = input_image[(y - 1) * width + (x - 1)];
        }
    }

    // Top-right corner
    if (threadIdx.x == TILE_SIZE - 1 && threadIdx.y == 0) {
        if(x < width - 1 && y > 0){
             shared_block[34 - 1] = input_image[(y - 1) * width + (x + 1)];
        }
    }

    // Bottom-left corner
    if (threadIdx.x == 0 && threadIdx.y == TILE_SIZE - 1) {
        if(x > 0 && y < height - 1){
            shared_block[(34 - 1) * 34] = input_image[(y + 1) * width + (x - 1)];
        }
    }

    //Bottom-right corner
    if (threadIdx.x == TILE_SIZE - 1 && threadIdx.y == TILE_SIZE - 1) {
        if(x < width - 1 && y < height - 1){
           shared_block[(34 - 1) * 34 + 34 - 1] = input_image[(y + 1) * width + (x + 1)];

        }
    }


    // Edges

    // Left edge
    if (threadIdx.x == 0 && x > 0) {
        shared_block[shared_y * 34] = input_image[y * width + (x - 1)];
    }

    // Right edge
    if (threadIdx.x == TILE_SIZE - 1 && x < width - 1) {
        shared_block[shared_y * 34 + 34 - 1] = input_image[y * width + (x + 1)];
    }

    // Top edge
    if (threadIdx.y == 0 && y > 0) {
        shared_block[shared_x] = input_image[(y - 1) * width + x];
    }
    
    //Bottom edge
    if (threadIdx.y == TILE_SIZE - 1 && y < height - 1) {
        shared_block[(34 - 1) * 34 + shared_x] = input_image[(y + 1) * width + x];
    }

  
    __syncthreads();

    // Apply the filter
    if (x < width && y < height) {
        int filter[filterWidth][filterHeight] = {
            {1, 1, 1},
            {1, 3, 1},
            {1, 1, 1}
        };

        int filter_sum = 0;
        for (int i = 0; i < filterWidth; i++) {
            for (int j = 0; j < filterHeight; j++) {
                filter_sum += filter[i][j];
            }
        }

        int sum = 0;
        for (int i = -1; i <= 1; i++) {
            for (int j = -1; j <= 1; j++) {
                sum += shared_block[(shared_y + i) * 34 + (shared_x + j)] * filter[i + 1][j + 1];
            }
        }

        output_image[y * width + x] = sum / filter_sum;
    }
}

int main() {

    // Image dimensions
    unsigned int height = 1024;
    unsigned int width = 1920;
    size_t size = height * width * sizeof(unsigned int);

    // Allocate memory for images on the host
    unsigned int* h_input_image = (unsigned int*)malloc(size);
    unsigned int* h_output_image = (unsigned int*)malloc(size);

    if (h_input_image == NULL || h_output_image == NULL) {
        fprintf(stderr, "Failed to allocate memory on host.\n");
        exit(1);
    }

    // Initialize the input image from file
    FILE* file = fopen("original_0.csv", "r");
    if (file == NULL) {
        fprintf(stderr, "Failed to open input file.\n");
        exit(1);
    }

    char line[10240];
    unsigned int row = 0;

    while (fgets(line, sizeof(line), file) && row < height) {
        char *token;
        unsigned int col = 0;

        token = strtok(line, ",");
        while (token != NULL && col < width) {
            h_input_image[row * width + col] = atoi(token);  // Use atoi to convert string to int
            token = strtok(NULL, ",");
            col++;
        }
        row++;
    }

    fclose(file);

    // Initialize the output image
    for (unsigned int i = 0; i < height; i++) {
        for (unsigned int j = 0; j < width; j++) {
            h_output_image[i * width + j] = 0;
        }
    }

    // Allocate memory for images on the device
    unsigned int* d_input_image;
    unsigned int* d_output_image;
    hipError_t cudaStatus;

    cudaStatus = hipMalloc(&d_input_image, size);
    checkCUDAError(cudaStatus, "hipMalloc failed for input image");

    cudaStatus = hipMalloc(&d_output_image, size);
    checkCUDAError(cudaStatus, "hipMalloc failed for output image");

    // Copy input image from host to device
    cudaStatus = hipMemcpy(d_input_image, h_input_image, size, hipMemcpyHostToDevice);
    checkCUDAError(cudaStatus, "hipMemcpy failed from host to device");

    // Define block and grid sizes
    dim3 blockSize(TILE_SIZE, TILE_SIZE);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Measure execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start recording
    hipEventRecord(start);

    // Launch filter kernel
    filter<<<gridSize, blockSize>>>(d_input_image, d_output_image, height, width);

    cudaStatus = hipGetLastError();
    checkCUDAError(cudaStatus, "Kernel launch failed");

    // Stop recording
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float microseconds = milliseconds * 1000.0f; // Convert milliseconds to microseconds
    float nanoseconds = microseconds * 1000.0f; // Convert microseconds to nanoseconds
    printf("Execution time: %f nanoseconds\n", nanoseconds);

    // Copy output image from device to host
    cudaStatus = hipMemcpy(h_output_image, d_output_image, size, hipMemcpyDeviceToHost);
    checkCUDAError(cudaStatus, "hipMemcpy failed from device to host");

    // Save output image to CSV file
    FILE* outFile = fopen("result.csv", "w");
    if (outFile == NULL) {
        fprintf(stderr, "Failed to open output file.\n");
        exit(1);
    }

    for (unsigned int i = 0; i < height; i++) {
        for (unsigned int j = 0; j < width; j++) {
            fprintf(outFile, "%d", h_output_image[i * width + j]);
            if (j < width - 1) {
                fprintf(outFile, ",");
            }
        }
        fprintf(outFile, "\n");
    }

    fclose(outFile);

    // Free device memory
    hipFree(d_input_image);
    hipFree(d_output_image);

    // Free host memory
    free(h_input_image);
    free(h_output_image);

    return 0;
}
